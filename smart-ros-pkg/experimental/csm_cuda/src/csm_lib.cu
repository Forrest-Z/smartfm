#include "hip/hip_runtime.h"
#include <stdio.h>
#include "algorithm"
#include "vector"
#include "iostream"
#include "math.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <csm_cuda/cudaVarType.h>
#include <hip/hip_math_constants.h>
#define PRINT_DEBUG 0
using namespace std;
using namespace thrust;
texture<int> tex_dev_voronoi_data_;
texture<float> tex_dev_free_space_data_;
bool use_free_space_=false;
bool got_normal_;
#define stream_number 4
hipStream_t stream_array[stream_number];
struct poseResult{
  double x,y,r,score;
};

__global__ void jfaKernel(int step, int voronoi_width, int voronoi_height, cudaPointNormal *p, int *voronoi_data){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if(i>=0 && j>=0 && i<voronoi_width && j<voronoi_height){
    int k=step;
    int offset = i + j*voronoi_width;
    int id = voronoi_data[offset];
    int nearest_id_candidates[9];
    int total_candidate = 0;
    if(id>0){
      nearest_id_candidates[total_candidate++] = id;
    }
    for(int step_x = i-k; step_x < i+k+1; step_x += k){
      for(int step_y = j-k; step_y < j+k+1; step_y += k){
	if(step_x < 0 || step_y < 0 || step_x >= voronoi_width || step_y >= voronoi_height)
	  continue;
	int offset_temp = step_x + step_y * voronoi_width;
	int nearest_seed = voronoi_data[offset_temp];
	if(nearest_seed > 0) 
	  nearest_id_candidates[total_candidate++] = nearest_seed;
      }
    }
    if(total_candidate > 0){
      int nearest_x = p[nearest_id_candidates[0]-1].x;
      int nearest_y = p[nearest_id_candidates[0]-1].y;
      
      int dist_nearest_x = nearest_x-i;
      int dist_nearest_y = nearest_y-j;
      int dist_nearest = dist_nearest_x*dist_nearest_x + dist_nearest_y*dist_nearest_y;
      int nearest_id_current = nearest_id_candidates[0];
      for(size_t seed_i=1; seed_i<total_candidate; seed_i++){
	nearest_x = p[nearest_id_candidates[seed_i]-1].x;
	nearest_y = p[nearest_id_candidates[seed_i]-1].y;
	dist_nearest_x = nearest_x-i;
	dist_nearest_y = nearest_y-j;
	int dist_nearest_cur = dist_nearest_x*dist_nearest_x + dist_nearest_y*dist_nearest_y;
	if(dist_nearest_cur < dist_nearest){
	  nearest_id_current = nearest_id_candidates[seed_i];
	  dist_nearest = dist_nearest_cur;
	}
      }
      voronoi_data[offset] = nearest_id_current;
    }
  }
}
__global__ void translationKernel(float res, float step_size, int voronoi_width, int voronoi_height, 
				 int x_step, int y_step, int x_range, int match_size,
				 int y_range, cudaPointNormal* match_p, 
				 cudaPointNormal* p, float* scores, bool use_free_space, bool use_normal){
  //check index
  //single thread dimension with xy grid for each offset
  //maximum thread is limited to 1024, hence need to call this function repeatedly
  //get the real offset in pixel
  //the range will be -range to range at step interval
  int offset_x = -x_range/2 + x_step * blockIdx.x;
  int offset_y = -y_range/2 + y_step * blockIdx.y;
  
  extern __shared__ float scores_per_block[];
  int match_idx = threadIdx.x;
  scores_per_block[match_idx] = 0.0;
  if(match_idx < match_size){
    //get pixel location of the matching xy
    int temp_x = match_p[match_idx].x;
    int temp_y = match_p[match_idx].y;
    int x = temp_x + offset_x;
    int y = temp_y + offset_y;
    if(x < voronoi_width && x >= 0 && y < voronoi_height && y >= 0){
      int grid_idx = x+y*voronoi_width;
      float free_space_data = 1.0;
      if(use_free_space)
	free_space_data = tex1Dfetch(tex_dev_free_space_data_, grid_idx);
      int nearest_seed = tex1Dfetch(tex_dev_voronoi_data_, grid_idx)-1;
      int nearest_x = p[nearest_seed].x;
      int nearest_y = p[nearest_seed].y;
      int dist_nearest_x = nearest_x - x;
      int dist_nearest_y = nearest_y - y;
      int dist_nearest_pixel = (int)(sqrtf((dist_nearest_x*dist_nearest_x + dist_nearest_y*dist_nearest_y)*res)/step_size);
      float dist_nearest = dist_nearest_pixel * step_size;
      float angle_score = 1.0;
      if(use_normal){
	if(dist_nearest > 0.3) angle_score = 0;
	else {
	  float a_x = p[nearest_seed].normal_x;
	  float a_y = p[nearest_seed].normal_y;
	  float b_x = match_p[match_idx].normal_x;
	  float b_y = match_p[match_idx].normal_y;
	  float scalar_product = a_x*b_x + a_y*b_y;
	  float mag = sqrt(a_x*a_x + a_y*a_y) * sqrt(b_x*b_x + b_y*b_y);
	  float cos_angle = scalar_product/mag;
	  float angle = acos(cos_angle);
	  angle_score = (HIP_PI_F - angle)/HIP_PI_F;
	}
      }
      scores_per_block[match_idx] = .7 * expf(-0.3*dist_nearest)*free_space_data + .3 * angle_score;
    }
  }
  
  __syncthreads();
  int nTotalThreads = blockDim.x;
  int thread2;
  while(nTotalThreads > 1){
    int halfPoint = (nTotalThreads >> 1);
    
    if(threadIdx.x < halfPoint){
      thread2 = threadIdx.x + halfPoint;
      

      if(thread2 < blockDim.x){
	scores_per_block[threadIdx.x] += scores_per_block[thread2];
      }
    }
    __syncthreads();
    
    nTotalThreads = halfPoint;
  }
  if(threadIdx.x == 0){
    int score_idx = blockIdx.x + blockIdx.y * gridDim.x; 
    scores[score_idx] += scores_per_block[0];
  }
}

void setDevFreeSpaceData(device_vector<float> &device_free_space){
  hipBindTexture(NULL, tex_dev_free_space_data_, raw_pointer_cast(device_free_space.data()),
		  device_free_space.size()*sizeof(float));
  use_free_space_ = true;
}

//form a euclidean based voronoi and return a device pointer of the resultant img
host_vector<int> voronoi_jfa(int voronoi_width, int voronoi_height, 
		 vector<cudaPointNormal> point,
		 device_vector<cudaPointNormal> &dev_p,
		 device_vector<int> &dev_voronoi_data, bool gotNormal
		){
  got_normal_ = gotNormal;
  if(PRINT_DEBUG)
    cout<<"Received pts "<<point.size()<<endl;
  host_vector<int> voronoi_data(voronoi_width*voronoi_height);
  for(size_t i=0; i<point.size(); i++)
    voronoi_data[point[i].x + point[i].y*voronoi_width] = i+1;
  dev_p = device_vector<cudaPointNormal>(point);
  dev_voronoi_data = device_vector<int>(voronoi_data);
  cudaPointNormal *dev_p_ptr = raw_pointer_cast(dev_p.data());
  int *dev_voronoi_data_ptr = raw_pointer_cast(dev_voronoi_data.data());
  dim3 grids(ceil(voronoi_width/16.f),ceil(voronoi_height/16.f));
  dim3 threads(16,16);
  for(int k=512; k>0; k/=2)
    jfaKernel<<<grids,threads>>>(k, voronoi_width, voronoi_height, dev_p_ptr, dev_voronoi_data_ptr);
  voronoi_data = dev_voronoi_data;
  
  hipBindTexture(NULL, tex_dev_voronoi_data_, dev_voronoi_data_ptr, dev_voronoi_data.size()*sizeof(int));
  for(int i=0; i<stream_number; i++) hipStreamCreate(&stream_array[i]);
  return voronoi_data;
}

poseResult best_translation(float resolution, float step_size, int x_step, int y_step, int x_range, int y_range,
  vector<cudaPointNormal> point,
  int voronoi_width, int voronoi_height,
  device_vector<cudaPointNormal> &dev_p,
  device_vector<int> &dev_voronoi_data, int stream_idx
		    )
{
  //step size will determine the resoution used when calculating
  //the distance value
  //Essentially the real distance will be passed through a step function
  //where each height of the step depends on the stepping resolution.
  //This helps to over estimate the cost function to ensure no
  //local minima
  
  //range from -_range to _range
  x_range *= 2;
  y_range *= 2;
  int total_scores = x_range/x_step * y_range/y_step;
  if(PRINT_DEBUG)
  cout<<"Total scores to be generated "<<total_scores <<
  "(range:"<<x_range<<"x"<<y_range<<", step:"<<x_step<<"x"<<y_step<<")"<<endl;
  vector<float> host_scores(total_scores, 0.0);
  float *dev_scores_ptr;
  hipMalloc(&dev_scores_ptr, total_scores*sizeof(float));
  hipMemcpyAsync(dev_scores_ptr, &host_scores[0], total_scores*sizeof(float), hipMemcpyHostToDevice, stream_array[stream_idx]);
 // host_vector<float> host_scores;
  vector<cudaPointNormal> match_p;
  cudaPointNormal *dev_p_ptr = raw_pointer_cast(dev_p.data());
  cudaPointNormal *dev_match_ptr;
  hipMalloc(&dev_match_ptr, 1024*sizeof(cudaPointNormal));
  //float *dev_scores_ptr = raw_pointer_cast(dev_scores.data());
  
  for(int i=0; i<point.size(); i++){
    int match_size;
    if(point[i].x>=0 && point[i].y>=0 && point[i].x<voronoi_width && point[i].y<voronoi_height){
      match_p.push_back(point[i]);
    }
    else continue;
    match_size = match_p.size();
    if(match_size == 1024 || i == point.size()-1){
      hipMemcpyAsync(dev_match_ptr, &match_p[0], match_size*sizeof(cudaPointNormal), hipMemcpyHostToDevice, stream_array[stream_idx]);
      dim3 grids(x_range/x_step,y_range/y_step);
      dim3 threads(1024);
      if(PRINT_DEBUG){
	cout<<"Grid size of "<<grids.x<<"x"<<grids.y<<" initiated."<<endl;
	cout<<"Thread size of "<<match_size<<" used."<<endl;
      }
      translationKernel<<<grids, threads, threads.x*sizeof(float), stream_array[stream_idx]>>>
				    (resolution, step_size,
				      voronoi_width, voronoi_height, 
				    x_step, y_step, x_range, match_size,
				    y_range, dev_match_ptr, 
				    dev_p_ptr, dev_scores_ptr, use_free_space_, got_normal_);
      match_p.clear();
    }
  }
  //hipDeviceSynchronize();
  hipMemcpyAsync(&host_scores[0], dev_scores_ptr, total_scores*sizeof(float), hipMemcpyDeviceToHost, stream_array[stream_idx]);
  int max_x, max_y;
  float max_value=0;
  
  for(int i=0; i<x_range/x_step; i++){
    //if(i>5) continue;
    for(int j=0; j<y_range/y_step; j++)
    {
      float temp_score = host_scores[i+j*x_range/x_step];
      if(PRINT_DEBUG)
	cout<<temp_score<<" ";
      if(max_value < temp_score){
	max_value = temp_score;
	max_x = i;
	max_y = j;
      }
    }
    if(PRINT_DEBUG)
      cout<<endl;
  }
  if(PRINT_DEBUG){
    cout<<"Best match: "<<(max_x-x_range/x_step/2)*x_step*resolution<<","<<(max_y-y_range/y_step/2)*y_step*resolution<<"("<<max_value/(double)point.size()*100<<")"<<endl;
    cout<<"Best match: "<<max_x<<","<<max_y<<"("<<max_value<<")"<<endl;
  }
  poseResult pose_result;
  pose_result.x = (max_x-x_range/x_step/2)*x_step*resolution;
  pose_result.y = (max_y-y_range/y_step/2)*y_step*resolution;
  pose_result.score = max_value/(double)point.size()*100;
  return pose_result;
}
