#include "hip/hip_runtime.h"
#include <stdio.h>
#include "algorithm"
#include "vector"
#include "iostream"
#include "math.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define PRINT_DEBUG 0
using namespace std;
using namespace thrust;
texture<int> tex_dev_voronoi_data_;
#define stream_number 4
hipStream_t stream_array[stream_number];
struct poseResult{
  double x,y,r,score;
};

__global__ void jfaKernel(int step, int voronoi_width, int voronoi_height, int *px, int *py, int *voronoi_data){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if(i>=0 && j>=0 && i<voronoi_width && j<voronoi_height){
    int k=step;
    int offset = i + j*voronoi_width;
    int id = voronoi_data[offset];
    int nearest_id_candidates[9];
    int total_candidate = 0;
    if(id>0){
      nearest_id_candidates[total_candidate++] = id;
    }
    for(int step_x = i-k; step_x < i+k+1; step_x += k){
      for(int step_y = j-k; step_y < j+k+1; step_y += k){
	if(step_x < 0 || step_y < 0 || step_x >= voronoi_width || step_y >= voronoi_height)
	  continue;
	int offset_temp = step_x + step_y * voronoi_width;
	int nearest_seed = voronoi_data[offset_temp];
	if(nearest_seed > 0) 
	  nearest_id_candidates[total_candidate++] = nearest_seed;
      }
    }
    if(total_candidate > 0){
      int nearest_x = px[nearest_id_candidates[0]-1];
      int nearest_y = py[nearest_id_candidates[0]-1];
      
      int dist_nearest_x = nearest_x-i;
      int dist_nearest_y = nearest_y-j;
      int dist_nearest = dist_nearest_x*dist_nearest_x + dist_nearest_y*dist_nearest_y;
      int nearest_id_current = nearest_id_candidates[0];
      for(size_t seed_i=1; seed_i<total_candidate; seed_i++){
	nearest_x = px[nearest_id_candidates[seed_i]-1];
	nearest_y = py[nearest_id_candidates[seed_i]-1];
	dist_nearest_x = nearest_x-i;
	dist_nearest_y = nearest_y-j;
	int dist_nearest_cur = dist_nearest_x*dist_nearest_x + dist_nearest_y*dist_nearest_y;
	if(dist_nearest_cur < dist_nearest){
	  nearest_id_current = nearest_id_candidates[seed_i];
	  dist_nearest = dist_nearest_cur;
	}
      }
      voronoi_data[offset] = nearest_id_current;
    }
  }
}
__global__ void translationKernel(float res, float step_size, int voronoi_width, int voronoi_height, 
				 int x_step, int y_step, int x_range, int match_size,
				 int y_range, int* match_x, int* match_y, 
				 int* px, int* py, float* scores){
  //check index
  //single thread dimension with xy grid for each offset
  //maximum thread is limited to 1024, hence need to call this function repeatedly
  //get the real offset in pixel
  //the range will be -range to range at step interval
  int offset_x = -x_range/2 + x_step * blockIdx.x;
  int offset_y = -y_range/2 + y_step * blockIdx.y;
  
  extern __shared__ float scores_per_block[];
  int match_idx = threadIdx.x;
  scores_per_block[match_idx] = 0.0;
  if(match_idx < match_size){
    //get pixel location of the matching xy
    int temp_x = match_x[match_idx];
    int temp_y = match_y[match_idx];
    int x = temp_x + offset_x;
    int y = temp_y + offset_y;
    if(x < voronoi_width && x >= 0 && y < voronoi_height && y >= 0){
      int grid_idx = x+y*voronoi_width;
      int nearest_seed = tex1Dfetch(tex_dev_voronoi_data_, grid_idx)-1;
      int nearest_x = px[nearest_seed];
      int nearest_y = py[nearest_seed];
      int dist_nearest_x = nearest_x - x;
      int dist_nearest_y = nearest_y - y;
      int dist_nearest_pixel = (int)(sqrtf((dist_nearest_x*dist_nearest_x + dist_nearest_y*dist_nearest_y)*res)/step_size);
      float dist_nearest = dist_nearest_pixel * step_size;
      scores_per_block[match_idx] = expf(-0.3*dist_nearest);
    }
  }
  
  __syncthreads();
  int nTotalThreads = blockDim.x;
  int thread2;
  while(nTotalThreads > 1){
    int halfPoint = (nTotalThreads >> 1);
    
    if(threadIdx.x < halfPoint){
      thread2 = threadIdx.x + halfPoint;
      

      if(thread2 < blockDim.x){
	scores_per_block[threadIdx.x] += scores_per_block[thread2];
      }
    }
    __syncthreads();
    
    nTotalThreads = halfPoint;
  }
  if(threadIdx.x == 0){
    int score_idx = blockIdx.x + blockIdx.y * gridDim.x; 
    scores[score_idx] += scores_per_block[0];
  }
}

//form a euclidean based voronoi and return a device pointer of the resultant img
host_vector<int> voronoi_jfa(int voronoi_width, int voronoi_height, 
		 vector<int> point_x, vector<int> point_y,
		 device_vector<int> &dev_px, device_vector<int> &dev_py,
		 device_vector<int> &dev_voronoi_data
		){
  if(PRINT_DEBUG)
    cout<<"Received pts "<<point_x.size()<<"x"<<point_y.size()<<endl;
  host_vector<int> voronoi_data(voronoi_width*voronoi_height);
  for(size_t i=0; i<point_x.size(); i++)
    voronoi_data[point_x[i] + point_y[i]*voronoi_width] = i+1;
  dev_px = device_vector<int>(point_x);
  dev_py = device_vector<int>(point_y);
  dev_voronoi_data = device_vector<int>(voronoi_data);
  int *dev_px_ptr = raw_pointer_cast(dev_px.data());
  int *dev_py_ptr = raw_pointer_cast(dev_py.data());
  int *dev_voronoi_data_ptr = raw_pointer_cast(dev_voronoi_data.data());
  dim3 grids(ceil(voronoi_width/16.f),ceil(voronoi_height/16.f));
  dim3 threads(16,16);
  for(int k=512; k>0; k/=2)
    jfaKernel<<<grids,threads>>>(k, voronoi_width, voronoi_height, dev_px_ptr, dev_py_ptr, dev_voronoi_data_ptr);
  voronoi_data = dev_voronoi_data;
  
  hipBindTexture(NULL, tex_dev_voronoi_data_, dev_voronoi_data_ptr, dev_voronoi_data.size()*sizeof(int));
  for(int i=0; i<stream_number; i++) hipStreamCreate(&stream_array[i]);
  return voronoi_data;
}

poseResult best_translation(float resolution, float step_size, int x_step, int y_step, int x_range, int y_range,
  vector<int> point_x, vector<int> point_y,
  int voronoi_width, int voronoi_height,
  device_vector<int> &dev_px, device_vector<int> &dev_py,
  device_vector<int> &dev_voronoi_data, int stream_idx
		    )
{
  //step size will determine the resoution used when calculating
  //the distance value
  //Essentially the real distance will be passed through a step function
  //where each height of the step depends on the stepping resolution.
  //This helps to over estimate the cost function to ensure no
  //local minima
  
  //range from -_range to _range
  x_range *= 2;
  y_range *= 2;
  int total_scores = x_range/x_step * y_range/y_step;
  if(PRINT_DEBUG)
  cout<<"Total scores to be generated "<<total_scores <<
  "(range:"<<x_range<<"x"<<y_range<<", step:"<<x_step<<"x"<<y_step<<")"<<endl;
  vector<float> host_scores(total_scores, 0.0);
  float *dev_scores_ptr;
  hipMalloc(&dev_scores_ptr, total_scores*sizeof(float));
  hipMemcpyAsync(dev_scores_ptr, &host_scores[0], total_scores*sizeof(float), hipMemcpyHostToDevice, stream_array[stream_idx]);
 // host_vector<float> host_scores;
  vector<int> match_x, match_y;
  int *dev_px_ptr = raw_pointer_cast(dev_px.data());
  int *dev_py_ptr = raw_pointer_cast(dev_py.data());
  int *dev_match_x_ptr, *dev_match_y_ptr;
  hipMalloc(&dev_match_x_ptr, 1024*sizeof(int));
  hipMalloc(&dev_match_y_ptr, 1024*sizeof(int));
  //float *dev_scores_ptr = raw_pointer_cast(dev_scores.data());
  

  
  for(int i=0; i<point_x.size(); i++){
    int match_size;
    if(point_x[i]>=0 && point_y[i]>=0 && point_x[i]<voronoi_width && point_y[i]<voronoi_height){
      match_x.push_back(point_x[i]);
      match_y.push_back(point_y[i]);
    }
    else continue;
    match_size = match_x.size();
    if(match_size == 1024 || i == point_x.size()-1){
      hipMemcpyAsync(dev_match_x_ptr, &match_x[stream_idx], match_size*sizeof(int), hipMemcpyHostToDevice, stream_array[stream_idx]);
      hipMemcpyAsync(dev_match_y_ptr, &match_y[stream_idx], match_size*sizeof(int), hipMemcpyHostToDevice, stream_array[stream_idx]);
      dim3 grids(x_range/x_step,y_range/y_step);
      dim3 threads(1024);
      if(PRINT_DEBUG){
	cout<<"Grid size of "<<grids.x<<"x"<<grids.y<<" initiated."<<endl;
	cout<<"Thread size of "<<match_size<<" used."<<endl;
      }
      translationKernel<<<grids, threads, threads.x*sizeof(float), stream_array[stream_idx]>>>
				    (resolution, step_size,
				      voronoi_width, voronoi_height, 
				    x_step, y_step, x_range, match_size,
				    y_range, dev_match_x_ptr, dev_match_y_ptr, 
				    dev_px_ptr, dev_py_ptr, dev_scores_ptr);
      match_x.clear();
      match_y.clear();
    }
  }
  //hipDeviceSynchronize();
  hipMemcpyAsync(&host_scores[0], dev_scores_ptr, total_scores*sizeof(float), hipMemcpyDeviceToHost, stream_array[stream_idx]);
  int max_x, max_y;
  float max_value=0;
  
  for(int i=0; i<x_range/x_step; i++){
    //if(i>5) continue;
    for(int j=0; j<y_range/y_step; j++)
    {
      float temp_score = host_scores[i+j*x_range/x_step];
      //cout<<temp_score<<" ";
      if(max_value < temp_score){
	max_value = temp_score;
	max_x = i;
	max_y = j;
      }
    }
    //cout<<endl;
  }
  if(PRINT_DEBUG){
    cout<<"Best match: "<<(max_x-x_range/x_step/2)*x_step*resolution<<","<<(max_y-y_range/y_step/2)*y_step*resolution<<"("<<max_value/(double)point_x.size()*100<<")"<<endl;
    cout<<"Best match: "<<max_x<<","<<max_y<<"("<<max_value<<")"<<endl;
  }
  poseResult pose_result;
  pose_result.x = (max_x-x_range/x_step/2)*x_step*resolution;
  pose_result.y = (max_y-y_range/y_step/2)*y_step*resolution;
  pose_result.score = max_value/(double)point_x.size()*100;
  return pose_result;
}
