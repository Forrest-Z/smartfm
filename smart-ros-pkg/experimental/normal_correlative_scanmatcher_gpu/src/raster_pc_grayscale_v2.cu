#include "hip/hip_runtime.h"
#include "handle.h"
#include "cpu_bitmap_grayscale.h"
#include "fm_stopwatch.h"
#include "algorithm"
#include "vector"
#include "iostream"
#include "math.h"

#include <thrust/device_vector.h>
#include <thrust/sort.h>

#define PI 3.14159265359
using namespace std;

__global__ void initImg(unsigned char *ptr, int size, unsigned char init_value=0){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    if(offset < size) {
        ptr[offset] =init_value;
    }
}
__global__ void initImg(int *ptr, int size, int init_value=0){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    if(offset < size) {
        ptr[offset] =init_value;
    }
}
__global__ void initImg(float *ptr, int size, float init_value=0){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    if(offset < size) {
        ptr[offset] =init_value;
    }
}

__global__ void drawTemplatePixelBased(float *data_x, float *data_y, 
				       float min_x, float min_y, float res, 
				       float *ptr, int *ptr_index, 
				       int img_width, int img_height){
  int pt_x = threadIdx.x + blockIdx.x * blockDim.x;
  int pt_y = threadIdx.y + blockIdx.y * blockDim.y;
  
  if(pt_x < img_width && pt_y <img_height){
    float x = pt_x*res+min_x;
    float y = pt_y*res+min_y;
    
    int i=blockIdx.z;
    
    float raw_data_x = data_x[i] - x;
    float raw_data_y = data_y[i] - y;
    float dist = raw_data_x*raw_data_x+raw_data_y*raw_data_y;
    
    
    int offset = pt_x + pt_y * img_width;
    if(ptr[offset] > dist) {
      ptr[offset] = dist;
      ptr_index[offset] = i;
    }
  }
}
__global__ void matchTemplateDirect(
  				    /************template data***************/
				    int *data_idx, float *data_x_template, float *data_y_template,
				    float min_x, float min_y, float max_x, float max_y,
				    int img_width, int img_height,
				    /************matching data**************/
				    float *data_x, float *data_y, int data_size, 
				    /**********output scores data***********/
				    float *scores, float *score_x, float *score_y, int *score_rot,
				    int x_size, int y_size, 
				    /**********matching spec****************/
				    float *x_steps, float *y_steps, float res,
				    /***********cached data*****************/
				    float *sin_data, float *cos_data
				    
 				  ){
  int offset_x = threadIdx.x + blockIdx.x*blockDim.x;
  int offset_y = threadIdx.y + blockIdx.y*blockDim.y;
  int offset_rot = threadIdx.z + blockIdx.z*blockDim.z;
  if(offset_x<x_size && offset_y<y_size && offset_rot < 180){
    int score_ptr = offset_x+offset_y*x_size+offset_rot*y_size*x_size;
    float score = 0;
    float sin = sin_data[offset_rot];
    float cos = cos_data[offset_rot];
    float center_offset_x =x_steps[offset_x];
    float center_offset_y =y_steps[offset_y];
    float img_width_p2 = img_width/2.f;
    float img_height_p2 = img_height/2.f;
    for(int i=0; i<data_size; i++){
      float pt_x_f = data_x[i];
      float pt_y_f = data_y[i];
      float ref_pt_x_f = pt_x_f * cos + pt_y_f * sin;
      float ref_pt_y_f = pt_y_f * cos - pt_x_f * sin;
      ref_pt_x_f += center_offset_x;
      ref_pt_y_f += center_offset_y;
      if(ref_pt_x_f < max_x && ref_pt_x_f > min_x && ref_pt_y_f < max_y && ref_pt_y_f > min_y)
      { 
	int ref_pt_x = roundf((ref_pt_x_f/res+img_width_p2));
	int ref_pt_y = roundf((ref_pt_y_f/res+img_height_p2));
	int ref_offset = ref_pt_x + ref_pt_y * img_width;
	int input_idx = data_idx[ref_offset];
	float x = data_x_template[input_idx];
	float y = data_y_template[input_idx];
	float dist_x = x - ref_pt_x_f;
	float dist_y = y - ref_pt_y_f;
	float d = sqrtf(dist_x * dist_x + dist_y * dist_y);
	score += expf(-d / 1.0);
      }
    }
    scores[score_ptr] = score;
    score_x[score_ptr] = center_offset_x;
    score_y[score_ptr] = center_offset_y;
    score_rot[score_ptr] = offset_rot;
  }
}

void findBestMatchDirect(
			  /************template data***************/
			  int *data_idx, float *data_x_template, float *data_y_template,
			  float min_x, float min_y, float max_x, float max_y,
			  int img_width, int img_height, int data_template_size,
			  /************matching data**************/
			  float *data_x, float *data_y, int data_size, 
			  /**********matching spec****************/
			  float x_range, float y_range, float res
			  ){
  hipEvent_t start, stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));
  int range_degree = 180;
  float cos_data[range_degree], sin_data[range_degree];
  for(int i=0; i<range_degree; i++){
    cos_data[i] = cos(PI/90*i);
    sin_data[i] = sin(PI/90*i);
  }
  //make sure all variables declared, now precalculating the range_x and range_y
  vector<float> x_steps, y_steps;
  for(float i=-x_range; i<=x_range; i+=res)	x_steps.push_back(i);
  for(float j=-y_range; j<=y_range; j+=res)	y_steps.push_back(j);
   
  int *dev_input;
  float *dev_data_x_template, *dev_data_y_template;
  float *dev_sin, *dev_cos, *dev_data_x, *dev_data_y, *dev_x_steps, *dev_y_steps;
  float *host_x_steps = &x_steps[0];
  float *host_y_steps = &y_steps[0];
  int size_of_scores = x_steps.size()*y_steps.size()*range_degree;
  cout<<"Size of scores = "<<size_of_scores<<endl;
  float *dev_scores;
  float *scores = new float[size_of_scores];
  float *dev_score_x, *dev_score_y;
  int *dev_score_rot;
  float *score_x = new float[size_of_scores];
  float *score_y = new float[size_of_scores];
  int *score_rot = new int[size_of_scores];
  
  HANDLE_ERROR(hipMalloc((void**)&dev_scores, size_of_scores*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_score_x, size_of_scores*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_score_y, size_of_scores*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_score_rot, size_of_scores*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_input, img_width*img_height*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_data_x_template, data_template_size*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_data_y_template, data_template_size*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_data_x, data_size*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_data_y, data_size*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_cos, range_degree*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_sin, range_degree*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_x_steps, x_steps.size()*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_y_steps, y_steps.size()*sizeof(float)));
  
  HANDLE_ERROR(hipMemcpy(dev_input, data_idx, img_width*img_height*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_data_x_template, data_x_template, data_template_size*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_data_y_template, data_y_template, data_template_size*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_data_x, data_x, data_size*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_data_y, data_y, data_size*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cos, cos_data, range_degree*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_sin, sin_data, range_degree*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_x_steps, host_x_steps, x_steps.size()*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_y_steps, host_y_steps, y_steps.size()*sizeof(float), hipMemcpyHostToDevice));
  
  int threadx = 8, thready = 8, threadrot = 4;
  dim3 threads(threadx,thready,threadrot);
  dim3 grids(ceil(x_steps.size()/(float)threadx), ceil(y_steps.size()/(float)thready), ceil(range_degree/(float)threadrot));
  std::cout<<"Running cuda with grid size of "<<grids.x<<","<<grids.y<<","<<grids.z<< " matching "<<data_size<<endl;
  
  matchTemplateDirect<<<grids, threads>>>(
  				    /************template data***************/
				    dev_input, dev_data_x_template, dev_data_y_template,
				    min_x, min_y, max_x, max_y,
				    img_width, img_height,
				    /************matching data**************/
				    dev_data_x, dev_data_y, data_size, 
				    /**********output scores data***********/
				    dev_scores, dev_score_x, dev_score_y, dev_score_rot,
				    x_steps.size(), y_steps.size(), 
				    /**********matching spec****************/
				    dev_x_steps, dev_y_steps, res,
				    /***********cached data*****************/
				    dev_sin, dev_cos
				    
 				  );
  HANDLE_ERROR(hipMemcpy(scores, dev_scores, size_of_scores*sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(score_x, dev_score_x, size_of_scores*sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(score_y, dev_score_y, size_of_scores*sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(score_rot, dev_score_rot, size_of_scores*sizeof(int), hipMemcpyDeviceToHost));
  MatchScoref ms;
  ms.score = 0;
  for(int i=0; i<size_of_scores; i++){
    if(ms.score < scores[i]){
      ms.score = scores[i];
      ms.x = score_x[i];
      ms.y = score_y[i];
      ms.rot = score_rot[i];
    }
  }
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elapsedTime;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
                                        start, stop));
  printf( "Time to generate:  %3.1f ms\n", elapsedTime);
  printf("best match %.2f, %.2f, %d, %f\n", ms.x, ms.y, ms.rot, ms.score/(data_size));
  
  
  
}

__global__ void rotateTemplate(int *input, float *data_x, float *data_y, int data_size, 
			       int img_width, int img_height, float *sin_data, float *cos_data, 
			       unsigned char *dev_bitmap, float range_cov, 
			       float min_x, float min_y, float max_x, float max_y, float res
			      ){
  int pt_x = threadIdx.x + blockIdx.x * blockDim.x;
  int pt_y = threadIdx.y + blockIdx.y * blockDim.y;
  if(pt_x < img_width && pt_y < img_height){
    float pt_x_f = pt_x - img_width/2.;
    float pt_y_f = pt_y - img_height/2.;

    int rot_idx = blockIdx.z;
    float sin = sin_data[rot_idx];
    float cos = cos_data[rot_idx];
 
    float ref_pt_x_f = pt_x_f * cos + pt_y_f * sin;
    float ref_pt_y_f = pt_y_f * cos - pt_x_f * sin;
    max_x/=res;
    max_y/=res;
    min_x/=res;
    min_y/=res;

    if(ref_pt_x_f < max_x && ref_pt_x_f > min_x && ref_pt_y_f < max_y && ref_pt_y_f > min_y)
    { 
      int ref_pt_x = roundf((ref_pt_x_f+img_width/2.));
      int ref_pt_y = roundf((ref_pt_y_f+img_height/2.));
      int ref_offset = ref_pt_x + ref_pt_y * img_width;
      int input_idx = input[ref_offset];
      if(input_idx < data_size && input_idx > 0){
        float x = data_x[input_idx];
        float y = data_y[input_idx];
        float dist_x = x - ref_pt_x_f*res;
        float dist_y = y - ref_pt_y_f*res;
        int offset = pt_x + (pt_y * img_width) + (img_width*img_height*rot_idx);
        
        float d = sqrtf(dist_x * dist_x + dist_y * dist_y);
        int gaussian_value = (int) (255 * expf(-d / range_cov));
        dev_bitmap[offset] = gaussian_value;
      }
    }
  }
}



CPUBitmap rotateMap(int *input, float *data_x, float *data_y, int data_size,
		    int img_width, int img_height, vector<float> &cos_data_vec, vector<float> &sin_data_vec,
		    float min_x, float min_y, float max_x, float max_y,
		    float res, float rot_res, float range_cov, unsigned char* dev_bitmap){
  hipEvent_t start, stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));
  
  
  int range_degree = cos_data_vec.size();
  //printf("Generating templates at %f deg with size of %d\n", rot_res, range_degree);
  float *cos_data = &cos_data_vec[0];
  float *sin_data = &sin_data_vec[0];
  int *dev_input;
  float *dev_sin, *dev_cos, *dev_data_x, *dev_data_y;
  
  
  CPUBitmap bitmap(img_width, img_height*range_degree);
  HANDLE_ERROR(hipMalloc((void**)&dev_input, img_width*img_height*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_data_x, data_size*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_data_y, data_size*sizeof(float)));
  
  HANDLE_ERROR(hipMalloc((void**)&dev_cos, range_degree*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_sin, range_degree*sizeof(float)));
  dim3 init_grid(ceil(img_width/16.f), ceil(img_height/16.f)*range_degree);
  initImg<<<init_grid, dim3(16,16)>>>(dev_bitmap, img_width*img_height*range_degree);
  HANDLE_ERROR(hipMemcpy(dev_input, input, img_width*img_height*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_data_x, data_x, data_size*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_data_y, data_y, data_size*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cos, cos_data, range_degree*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_sin, sin_data, range_degree*sizeof(float), hipMemcpyHostToDevice));
  
  dim3 grids(ceil(img_width/16.f), ceil(img_height/16.f), range_degree);
  dim3 threads(16,16);
  rotateTemplate<<<grids, threads>>>(dev_input, dev_data_x, dev_data_y, data_size, 
			       img_width, img_height, dev_sin, dev_cos, dev_bitmap, range_cov,
			       min_x, min_y, max_x, max_y, res
		);
  
  HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, img_width*img_height*range_degree, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elapsedTime;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
                                        start, stop));
  //printf( "Time to generate:  %3.1f ms\n", elapsedTime);
  HANDLE_ERROR( hipFree( dev_input ) );
  HANDLE_ERROR( hipFree( dev_data_x ) );
  HANDLE_ERROR( hipFree( dev_data_y ) );
  //HANDLE_ERROR( hipFree( dev_bitmap ) );
  HANDLE_ERROR( hipFree( dev_cos ) );
  HANDLE_ERROR( hipFree( dev_sin ) );
  bitmap.rot_size = range_degree;
  return bitmap;
}

void rasterMap(float *data_x, float *data_y, float min_x, float min_y, int data_size, int DIMW, int DIMH, 
		    float res, float *normals, int oversample_ratio, float *bitmap, int *nn_index){
  //capture the start time
  hipEvent_t start, stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));
  
  //let's create a template consists of 2 degree turning for now
  int range_degree = 1;
  //declare device variable
  float *dev_data_x, *dev_data_y, *dev_cos, *dev_sin;
  float *dev_bitmap;
  int *dev_nn_index;
  
 
 
  //allocation of GPU memory
  HANDLE_ERROR(hipMalloc((void**)&dev_data_x, data_size*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_data_y, data_size*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_cos, range_degree*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_sin, range_degree*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, DIMW*DIMH*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_nn_index, DIMW*DIMH*sizeof(int)));
  
  dim3    grids(ceil(DIMW/16.f),ceil(DIMH*range_degree/16.f));
  dim3    threads(16,16);
  initImg<<<grids,threads>>>( dev_bitmap, DIMW*DIMH*range_degree, DIMW*DIMW+DIMH*DIMH);
  initImg<<<grids,threads>>>(dev_nn_index, DIMW*DIMH, data_size);
  //copy data to the allocated GPU memory
  HANDLE_ERROR(hipMemcpy(dev_data_x, data_x, data_size*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_data_y, data_y, data_size*sizeof(float), hipMemcpyHostToDevice));
  float cos_data[range_degree], sin_data[range_degree];
  for(int i=0; i<range_degree; i++){
    cos_data[i] = cos(PI/90*i);
    sin_data[i] = sin(PI/90*i);
  }
  HANDLE_ERROR(hipMemcpy(dev_cos, cos_data, range_degree*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_sin, sin_data, range_degree*sizeof(float), hipMemcpyHostToDevice));
    
  //start GPU proccess, let's get each thread to compute a different rotation based on blockDim.y
  dim3 threadsPerGrid(16, 16);
  dim3 numBlocks(ceil(DIMW/16.f), ceil(DIMH/16.f), data_size);
  drawTemplatePixelBased<<<numBlocks, threadsPerGrid>>>(dev_data_x, dev_data_y, min_x, min_y, res, 
							  dev_bitmap, dev_nn_index, DIMW, DIMH);
  
  HANDLE_ERROR(hipMemcpy(bitmap, dev_bitmap, DIMW*DIMH*sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(nn_index, dev_nn_index, DIMW*DIMH*sizeof(int), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elapsedTime;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
                                        start, stop));
  //printf( "Time to generate:  %3.1f ms\n", elapsedTime);

 
  HANDLE_ERROR( hipFree( dev_nn_index ) );
  HANDLE_ERROR( hipFree( dev_data_x ) );
  HANDLE_ERROR( hipFree( dev_data_y ) );
  HANDLE_ERROR( hipFree( dev_bitmap ) );
  HANDLE_ERROR( hipFree( dev_cos ) );
  HANDLE_ERROR( hipFree( dev_sin ) );
  
}
texture<unsigned char> tex_dev_template_;  
__global__ void matchTemplate(unsigned char *match_template, int *data_x, int *data_y, int *input_data_count,
			      int data_size, float *scores, int *score_x, int *score_y,
			      int *score_rot, int img_width, int img_height,
			      int *x_steps, int *y_steps, int *r_steps,
			      int x_size, int y_size, int r_size,
			      int ini_r
 			    ){
  int offset_x = threadIdx.x + blockIdx.x*blockDim.x;
  int offset_y = threadIdx.y + blockIdx.y*blockDim.y;
  int offset_rot = threadIdx.z + blockIdx.z*blockDim.z;
  
  if(offset_x<x_size && offset_y<y_size && offset_rot < r_size){
    
    
    
    float score = 0;
    int score_ptr = offset_x+offset_y*x_size+offset_rot*y_size*x_size;
    
    //change the logic here to real value comparison.
    int center_offset_x =x_steps[offset_x];
    int center_offset_y =y_steps[offset_y];
    int center_offset_rot = r_steps[offset_rot];
    score_rot[score_ptr] = center_offset_rot;
    score_x[score_ptr] = center_offset_x;
    score_y[score_ptr] = center_offset_y;
    int offset_template = (img_width*img_height*center_offset_rot);
    for(int i=0; i<data_size; i++){
      int x = data_x[i]+center_offset_x;
      int y = data_y[i]+center_offset_y;
      if(x>=0 && y>=0 && x<img_width && y<img_height){
	int score_temp = match_template[x + y*img_width + offset_template];
	//int score_temp = tex1Dfetch(tex_dev_template_,x + y*img_width + offset_template);
	score += input_data_count[i]*score_temp;
      }
    }
    
    scores[score_ptr] = score;
  }
  return;
}
 
struct sortMatchScore{
  __host__ __device__
  bool operator()(MatchScore ms1, MatchScore ms2){
    return ms1.score > ms2.score;
  }
};

MatchScore findBestMatch(int *data_x, int *data_y, int *input_data_count, int data_size,
		   int raw_data_size, unsigned char *dev_template, int img_width, int img_height,
		   float res, float deg_res, vector<int> x_steps, vector<int> y_steps, vector<int> r_steps,
		   int ini_r
		  ){
  
  hipEvent_t start, stop, start_match, stop_match;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));
  HANDLE_ERROR(hipEventCreate(&start_match));
  HANDLE_ERROR(hipEventCreate(&stop_match));
  
  int rot_step = 360.f/deg_res;
  
  
  
 
  int *dev_data_x, *dev_data_y, *dev_input_data_count;
  //thrust::device_vector<MatchScore> dev_score_vec(x_step*y_step*4*rot_step);
  
  //MatchScore *dev_scores = thrust::raw_pointer_cast(&dev_score_vec[0]);
  int size_of_scores = x_steps.size()*y_steps.size()*r_steps.size();
  //cout<<"Size of scores = "<<size_of_scores<<endl;
  float *dev_scores;
  float *scores = new float[size_of_scores];
  int *dev_score_x, *dev_score_y, *dev_score_rot;
  int *score_x = new int[size_of_scores];
  int *score_y = new int[size_of_scores];
  int *score_rot = new int[size_of_scores];
  int *x_steps_array = &x_steps[0], *dev_x_steps;
  int *y_steps_array = &y_steps[0], *dev_y_steps;
  int *r_steps_array = &r_steps[0], *dev_r_steps;
  HANDLE_ERROR(hipMalloc((void**)&dev_scores, size_of_scores*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_score_x, size_of_scores*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_score_y, size_of_scores*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_score_rot, size_of_scores*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_data_x, data_size*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_data_y, data_size*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_x_steps, x_steps.size()*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_y_steps, y_steps.size()*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_r_steps, r_steps.size()*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_input_data_count, data_size*sizeof(int)));
  HANDLE_ERROR(hipMemcpy(dev_data_x, data_x, data_size*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_data_y, data_y, data_size*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_input_data_count, input_data_count, data_size*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_x_steps, x_steps_array, x_steps.size()*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_y_steps, y_steps_array, y_steps.size()*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_r_steps, r_steps_array, r_steps.size()*sizeof(int), hipMemcpyHostToDevice));
 // HANDLE_ERROR(hipBindTexture(NULL, tex_dev_template_, dev_template, img_width*img_height));
  int threadx = 8, thready = 8, threadrot = 4;
  dim3 threads(threadx,thready,threadrot);
  dim3 grids(ceil(x_steps.size()/(float)threadx), ceil(y_steps.size()/(float)thready), ceil(r_steps.size()/(float)threadrot));
  //std::cout<<"Running cuda with grid size of "<<grids.x<<","<<grids.y<<","<<grids.z<< " matching "<<data_size<<endl;
  HANDLE_ERROR(hipEventRecord(start_match, 0));
  matchTemplate<<<grids, threads>>>(dev_template, dev_data_x, dev_data_y, dev_input_data_count,
				    data_size, dev_scores, dev_score_x, dev_score_y, 
				    dev_score_rot, img_width, img_height/rot_step,
				    dev_x_steps, dev_y_steps, dev_r_steps,
				    x_steps.size(), y_steps.size(), r_steps.size(), ini_r
				   );
  HANDLE_ERROR(hipEventRecord(stop_match,0));
  HANDLE_ERROR(hipEventSynchronize(stop_match));
  float elapsedTimeCore;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTimeCore,
                                        start_match, stop_match));
  
  //printf( "Time for match_core:  %3.1f ms\n", elapsedTimeCore);
  //thrust::sort(dev_score_vec.begin(), dev_score_vec.end(), sortMatchScore());
  HANDLE_ERROR(hipMemcpy(scores, dev_scores, size_of_scores*sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(score_x, dev_score_x, size_of_scores*sizeof(int), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(score_y, dev_score_y, size_of_scores*sizeof(int), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(score_rot, dev_score_rot, size_of_scores*sizeof(int), hipMemcpyDeviceToHost));
  MatchScore ms;
  
  thrust::device_vector<float> d_scores(scores, scores+size_of_scores);
  thrust::device_vector<float>::iterator iter =
  thrust::max_element(d_scores.begin(), d_scores.end());

  unsigned int position = iter - d_scores.begin();
  float max_val = *iter;

  //std::cout << "The maximum value is " << max_val << " at position " << position << std::endl;
  ms.score = scores[position];
  ms.x = score_x[position];
  ms.y = score_y[position];
  ms.rot = score_rot[position];
  /*ms.score = 0;
  for(int i=0; i<size_of_scores; i++){
    if(ms.score < scores[i]){
      ms.score = scores[i];
      ms.x = score_x[i];
      ms.y = score_y[i];
      ms.rot = score_rot[i];
    }
  }*/
  
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elapsedTime;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
                                        start, stop));
  
  //printf( "Time to match:  %3.1f ms\n", elapsedTime);
  //the following line caused thrust::system::system_error, invalid device pointer
//  hipDeviceReset();
  
  free(scores);
  free(score_x);
  free(score_y);
  free(score_rot);
  HANDLE_ERROR(hipFree(dev_scores));
  HANDLE_ERROR(hipFree(dev_score_x));
  HANDLE_ERROR(hipFree(dev_score_y));
  HANDLE_ERROR(hipFree(dev_score_rot));
  HANDLE_ERROR(hipFree(dev_data_x));
  HANDLE_ERROR(hipFree(dev_data_y));
  HANDLE_ERROR(hipFree(dev_x_steps));
  HANDLE_ERROR(hipFree(dev_y_steps));
  HANDLE_ERROR(hipFree(dev_r_steps));
  HANDLE_ERROR(hipFree(dev_input_data_count));
  return ms;
}
